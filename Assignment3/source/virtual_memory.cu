#include "hip/hip_runtime.h"
﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void init_invert_page_table(VirtualMemory *vm) {

  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    vm->invert_page_table[i] = 0x80000000; // invalid := MSB is 1     4KB
    vm->invert_page_table[i + vm->PAGE_ENTRIES] = i; //Store page number    4KB
    vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] = 0;  //Frequency clock, increase everytime, initial as 0    4KB
  }
}

__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES) {
  // init variables
  vm->buffer = buffer;
  vm->storage = storage;
  vm->invert_page_table = invert_page_table;
  vm->pagefault_num_ptr = pagefault_num_ptr;

  // init constants
  vm->PAGESIZE = PAGESIZE;
  vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
  vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
  vm->STORAGE_SIZE = STORAGE_SIZE;
  vm->PAGE_ENTRIES = PAGE_ENTRIES;

  // before first vm_write or vm_read
  init_invert_page_table(vm);
}

__device__ u32 paging(VirtualMemory *vm, u32 pageNum, u32 offset, u32 is_read) {
  //A clock, every read/write, the clock time will inrease 1
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    if (!vm->invert_page_table[i]) {
      vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] += 1;
    }
  }

  //Check wether the frame exita
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    // "0" represent valid
    if ((vm->invert_page_table[i + vm->PAGE_ENTRIES] == pageNum) && (!vm->invert_page_table[i])) {
      vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] = 0;  //Clear the frequency clock
      return i * vm->PAGESIZE + offset;
    }
  }

  
  //Check whether the corresponding frame is empty
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    // "1" represent invalid;
    if (vm->invert_page_table[i] >> 31) {
      vm->invert_page_table[i] = 0;   //Refresh the page table valid bit
      *(vm->pagefault_num_ptr) = *(vm->pagefault_num_ptr) + 1;

      vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] = 0;  //Clear the frequency clock
      return i * vm->PAGESIZE + offset;
    }
  }


  //If there is no empty space, finad the least recently used block to swap in
  u32 leastUsedPage = 0;    
  *(vm->pagefault_num_ptr) = *(vm->pagefault_num_ptr) + 1;
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    if (vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] > vm->invert_page_table[leastUsedPage + 2 * vm->PAGE_ENTRIES]) {
      leastUsedPage = i;
    }
  }
  //printf("leastUsedPage %d \n", leastUsedPage);
  //Swap in or swap out
  for (int i = 0; i < vm->PAGESIZE; i++) {
    u32 storageSwapAddr = vm->invert_page_table[leastUsedPage + vm->PAGE_ENTRIES] * vm->PAGESIZE + i;
    u32 swapFrame = leastUsedPage * vm->PAGESIZE + i;
    u32 storageAddr = pageNum * vm->PAGESIZE + i;
    if (is_read) {
      vm->storage[storageSwapAddr] = vm->buffer[swapFrame];     
      vm->buffer[swapFrame] = vm->storage[storageAddr];
    } else {
      vm->storage[storageSwapAddr] = vm->buffer[swapFrame];
    }
  }
  //Refresh the page table
  vm->invert_page_table[leastUsedPage + vm->PAGE_ENTRIES] = pageNum;
  vm->invert_page_table[leastUsedPage + 2 * vm->PAGE_ENTRIES] = 0; //Clear the frequency clock

  return leastUsedPage * vm->PAGESIZE + offset;
}



__device__ uchar vm_read(VirtualMemory *vm, u32 addr) {
  /* Complete vm_read function to read single element from data buffer */
  u32 pageNum = addr / vm->PAGESIZE;
  u32 offset = addr % vm->PAGESIZE;

  u32 address = paging(vm, pageNum, offset, 1);
  return vm->buffer[address];
}



__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value) {
  /* Complete vm_write function to write value into data buffer */
  u32 pageNum = addr / vm->PAGESIZE;
  u32 offset = addr % vm->PAGESIZE;
  
  u32 address = paging(vm, pageNum, offset, 0);
  vm->buffer[address] = value;
}


__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,
                            int input_size) {
  /* Complete snapshot function togther with vm_read to load elements from data
   * to result buffer */
  for (int i = 0; i < input_size; i++) {
    results[i] = vm_read(vm, i + offset);
  }
}


